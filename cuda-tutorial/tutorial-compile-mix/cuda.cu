
#include <hip/hip_runtime.h>
#include <numeric>
#include <iostream>


#define N 10000

__global__ void initialize(double *v)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	v[tid] = (tid + 1);
}


void cuda_stuff()
{
	double *dev_v;
	const size_t size = N*sizeof(double);
	
	hipMalloc(&dev_v, size);

	initialize<<<dim3(625), dim3(16)>>>(dev_v);

	double *vout = (double*) malloc(size);	
	hipMemcpy(vout, dev_v, size, hipMemcpyDeviceToHost);
	
	double sum = std::accumulate(vout, vout+N, 0);

	std::cout << "Sum: " << sum << " == "<< (N*(N+1)/2) << "\n";
	
	hipFree(dev_v);
	free(vout);
}
