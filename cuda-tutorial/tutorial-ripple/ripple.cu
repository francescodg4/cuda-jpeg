#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

#define DIM 256


__global__ void ripple(unsigned char *dev_img, int tick)
{
	/* ... */
}


int main()
{
	unsigned char *dev_img;
	
	hipMalloc(&dev_img, DIM*DIM);

	unsigned char *img = (unsigned char *) malloc(DIM*DIM*sizeof(unsigned char));
	
	for (int t = 0; t < 100; t++) {

		/* Kernel launch ripple(..., t) */
		
		hipMemcpy(img, dev_img, DIM*DIM, hipMemcpyDeviceToHost);

		// Save output to file
		std::string filename = "/tmp/ripple";
		filename += std::to_string(i);
		filename += ".gray";
		
		std::cout << filename << "\n";
			
		FILE *fp = fopen(filename.c_str(), "wb");

		fwrite(img, DIM*DIM, 1, fp);
		fclose(fp);
	}
	
	/* Cleanup */
	
	return 0;
}
